#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm> 

__global__ void mandelKernel(float gLowerX, float gLowerY,float gStepX, float gStepY, int gMaxIterations,int* gResult) 
{
    int a =blockIdx.x *blockDim.x + threadIdx.x;
    int i = a%1600;
    int j = a/1600;
    float x = gLowerX + i * gStepX;
    float y = gLowerY + j * gStepY;
    float z_re = x; 
    float z_im = y;
    float  temp;
    int k=0;
    while((z_re * z_re + z_im * z_im <= 4)&& (k < gMaxIterations))
    {

        temp = z_re * z_re - z_im * z_im+x;
        //new_im = 2.f * z_re * z_im;
        z_im = y + 2.f * z_re * z_im;
        z_re = temp;
        k++;
    }
         gResult[a]=k;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    //配置主機記憶體
    int *pResult;
    // pResult = (int *)malloc(resX*resY*sizeof(int));
    hipHostAlloc( (void**)&pResult,resX*resY*sizeof(int),hipHostMallocDefault);
    //配置顯示卡記憶體
    int  *gResult;
    size_t pitch;
    hipMallocPitch((void**)&gResult, &pitch,  1875 * sizeof(int), 1024);

    // //嘗試 zero copy
    // cudaHostGetDevicePointer(&gResult, pResult, 0);
    //cudaMallocPitch((void**)&gResult, &pitch,  resX*resY * sizeof(int), 1);
    //printf("%f %f %f %f %d\n",*pLowerX,*pLowerY,*pStepX,*pStepY,*pMaxIterations);
    //printf("%lu\n",pitch);
    dim3 threadsPerBlock(32);
    dim3 numBlocks(60000);
    mandelKernel<<<numBlocks,threadsPerBlock>>>(lowerX,lowerY,stepX,stepY,maxIterations,gResult);
    hipMemcpy(pResult, gResult, resX*resY*sizeof(int), hipMemcpyDeviceToHost);

    //給回img
    std::copy(pResult, pResult+1920000,img);

    // //釋放記憶體
    hipHostFree(pResult);
    hipFree(gResult);

}
